#include "hip/hip_runtime.h"
// free_list_test.cu
#include <HashedArrayTree.cu>
#include <gtest/gtest.h>

#include "MacroUtilities.cpp"

#include <cstddef>
#include <type_traits>
#define HANDLE_THREADS 8

template<typename T,typename L>
class IteratorTest : public ::testing::Test {
    protected:
     virtual void SetUp() {
		//create dummy container
	}
	typedef HashedArrayTree<T,L>			container;
	typedef typename container::iterator	iterator; 


	DEFINE(EmptyTest,		HANDLE_THREADS)
     DEFINE(BuddyOffSetTest, HANDLE_THREADS)
     DEFINE(CopyConstructorTest, HANDLE_THREADS)
     DEFINE(JoinOperatorTest, HANDLE_THREADS)
     DEFINE(DereferenceOperatorTest, HANDLE_THREADS)
     DEFINE(IndirectionOperatorTest, HANDLE_THREADS)
     DEFINE(EqualityComparableTest, HANDLE_THREADS)
     DEFINE(DefaultConstructionTest, HANDLE_THREADS)
     DEFINE(CopyAssignableTest, HANDLE_THREADS)
     DEFINE(NullablePointerTest, HANDLE_THREADS)
     DEFINE(BiderectionalTest, HANDLE_THREADS)
     DEFINE(RandomAccessTest, HANDLE_THREADS)
     DEFINE(VoidTest, HANDLE_THREADS)
     DEFINE(BoolConvertTest, HANDLE_THREADS)
     DEFINE(IfStatementTest, HANDLE_THREADS)
     DEFINE(ConvertToConstTest, HANDLE_THREADS)
     DEFINE(ConstDereferenceTest, HANDLE_THREADS)
     DEFINE(DeviceSingleTest, HANDLE_THREADS)
     DEFINE(DeviceMultipleTest, HANDLE_THREADS)
};
template<typename T,typename L>
void	IteratorTest<T,L>::EmptyTest(){
	iterator it_local;	
}

void IteratorTest::ConstDereferenceTest() {
/*
	const Handle<int> h(0, 0, base_ptr);
     *h;
*/
};

void IteratorTest::BuddyOffSetTest() {
/*
	int buddy = handle.buddy_offset();
     EXPECT_EQ(8, buddy);
*/
};

void IteratorTest::CopyConstructorTest() {
//    Handle<int> handle_new(handle);
};
void IteratorTest::JoinOperatorTest() {
/* 
	Handle<int> h_1(handle);
     Handle<int> h_2(handle);

     h_1._offset = 8;
     h_1.combine(h_2);
     EXPECT_EQ(h_1._offset, 0);
     EXPECT_EQ(h_1._size, 16);
*/
};
void IteratorTest::DereferenceOperatorTest() {
/* 
	*base = 0;
     EXPECT_EQ(*handle, 0);
     *base = 1;
     EXPECT_EQ(*handle, 1);
*/
};
void IteratorTest::IndirectionOperatorTest() {
/*
	Handle<int>* handle_ptr = &handle;
     int offset = handle_ptr->_offset;
     EXPECT_EQ(offset, 0);
*/
};
void IteratorTest::EqualityComparableTest() {
/*
	Handle<int> handle_2(0, 2, base_ptr);
     bool test_1 = (handle_2 == handle);
     EXPECT_TRUE(test_1);

     Handle<int> handle_3(1, 2, base_ptr);
     bool test_2 = (handle_3 == handle);
     EXPECT_FALSE(test_2);

     double* base_ptr2 = new double;
     Handle<double> handle_4(0, 8, base_ptr2);
     bool test_3 = (handle_4 == handle);
     EXPECT_FALSE(test_3);
*/
};
void IteratorTest::DefaultConstructionTest() {
//     Handle<int> handle_2();
};
void IteratorTest::CopyAssignableTest() {
/*     
	Handle<int> handle_2 = handle;
     bool test = (handle_2 == handle);
     EXPECT_TRUE(test);
*/
};
void IteratorTest::NullablePointerTest() {
/*     
	std::nullptr_t null;
     Handle<int> handle_2(null);
     bool test_1 = (handle_2 == null);
     EXPECT_TRUE(test_1);

     Handle<int> handle_3 = null;
     bool test_2 = (handle_3 == null);
     EXPECT_TRUE(test_2);

     EXPECT_FALSE(handle == null);
     EXPECT_FALSE(null == handle);
     EXPECT_TRUE(handle != null);
     EXPECT_TRUE(null != handle);
*/
};
void IteratorTest::BiderectionalTest() {
	

/*     
	Handle<int> handle_2 = handle;
     handle_2++;
     EXPECT_TRUE(handle_2 != handle);
     handle_2--;
     EXPECT_TRUE(handle_2 == handle);
     --handle_2;
     ++handle_2;
     *handle_2++;
     EXPECT_EQ(*handle_2, 1);
     *handle_2--;
     EXPECT_EQ(*handle_2, 0);
*/
};
void IteratorTest::RandomAccessTest() {
/*     
	Handle<int> handle_2 = handle;

     handle_2 += 1;
     EXPECT_EQ(*handle_2, 1);
     handle_2 -= 1;
     EXPECT_EQ(*handle_2, 0);

     Handle<int> handle_3 = handle + 1;
     EXPECT_EQ(*handle_3, 1);
     Handle<int> handle_4 = 1 + handle;
     EXPECT_EQ(*handle_4, 1);
     Handle<int> handle_5 = handle_4 - 1;
     EXPECT_EQ(*handle_5, 0);

     Handle<int> handle_6 = handle;
     handle_6 += 1;
     int dif = handle_6 - handle;
     EXPECT_EQ(dif, 1);
     EXPECT_TRUE(handle < handle_6);
     EXPECT_FALSE(handle > handle_6);
     EXPECT_TRUE(handle <= handle_6);
     EXPECT_FALSE(handle >= handle_6);

     EXPECT_EQ(handle[0], 0);
     EXPECT_EQ(handle[1], 1);
*/
};

void IteratorTest::VoidTest() { 
//	Handle_void handle_void; 
}

void IteratorTest::BoolConvertTest() { 
//	bool test = handle; 
}

void IteratorTest::IfStatementTest() {
/*     
     bool True = true;

     True ? handle : 0;
     !True ? handle : 0;
*/
}

void IteratorTest::ConvertToConstTest() {
/*     
	bool convertable = std::is_convertible<Handle<int>, const Handle<int>>::value;
     EXPECT_TRUE(convertable);
     convertable = std::is_convertible<Handle<int>, const Handle<int>>::value;
     EXPECT_TRUE(convertable);
*/
};

__global__ void transferS(Handle<int> x, Handle<int> y) {
     *y = *x;
};
void IteratorTest::DeviceSingleTest() {
/*
	int x = 2;
     int y = 0;

     int* x_d;
     int* y_d;

     hipMalloc((void**)&x_d, sizeof(int));
     hipMalloc((void**)&y_d, sizeof(int));

     hipMemcpy(x_d, &x, sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(y_d, &y, sizeof(int), hipMemcpyHostToDevice);

     Handle<int> x_h(0, 1, x_d);
     Handle<int> y_h(0, 1, y_d);
     transferS << <1, 1>>> (x_h, y_h);
     hipDeviceSynchronize();
     hipMemcpy(&y, y_d, sizeof(int), hipMemcpyDeviceToHost);
     EXPECT_EQ(x, y);
     hipFree(x_d);
     hipFree(y_d);
*/
}

__global__ void transferM(Handle<int> x, Handle<int> y) {
     y[1] = x[1];
};

void IteratorTest::DeviceMultipleTest() {
/*     
	int x[2] = {2, 3};
     int y[2] = {0, 1};

     int* x_d;
     int* y_d;
     int size = 2 * sizeof(int);

     hipMalloc((void**)&x_d, size);
     hipMalloc((void**)&y_d, size);
     hipMemcpy(x_d, &x, size, hipMemcpyHostToDevice);
     hipMemcpy(y_d, &y, size, hipMemcpyHostToDevice);

     Handle<int> x_h(0, 2, x_d);
     Handle<int> y_h(0, 2, y_d);
     transferM << <1, 1>>> (x_h, y_h);
     hipDeviceSynchronize();
     hipMemcpy(&y, y_d, size, hipMemcpyDeviceToHost);
     EXPECT_EQ(x[1], y[1]);
     EXPECT_FALSE(x[0] == y[0]);
     hipFree(x_d);
     hipFree(y_d);
*/
}


// clang-format off
// python:key:function=EmptyTest
// DeviceSingleTest DeviceMultipleTest ConstDereferenceTest ConvertToConstTest IfStatementTest BoolConvertTest VoidTest BuddyOffSetTest CopyConstructorTest JoinOperatorTest DereferenceOperatorTest IndirectionOperatorTest EqualityComparableTest DefaultConstructionTest CopyAssignableTest NullablePointerTest BiderectionalTest RandomAccessTest
// python:key:concurency=Single Threaded
// python:key:type=int float
// python:key:location=host device pinned unified
//
// python:template=TEST_F($IteratorTest<|type|,|location|>$,|function||concurency|){this->|function||concurency|();};
// python:start
// python:include=iterator.test
#include "iterator.test"
// python:end
// clang-format on
#undef HANDLE_THREADS
