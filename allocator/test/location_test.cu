#include <location.cu>
#include <gtest/gtest.h>

#define LOCATION_THREADS 8

#include <MacroUtilities.cpp>
#include <vector>
#include <thread>

#include<stdio.h>
#include<stdlib.h>

using namespace Flamingo::Memory;
// host location test
template <Region M>
class LocationTest : public ::testing::Test {
	protected:
	typedef int		test_type;
	typedef test_type*	pointer;
	pointer			h_ptr;
	pointer			d_ptr;
	int length=16;
	int size=sizeof(test_type)*length;
	location<M> policy;

	virtual void SetUp(){
		h_ptr=static_cast<pointer>(malloc(size));
		hipMalloc((void**)&d_ptr, size);
		for(int i=0; i<length; i++){
			h_ptr[i]=i; 
		}
		policy.MemCopy(h_ptr,d_ptr,size);
	}

	virtual void TearDown(){
		free(h_ptr);
		hipFree(d_ptr);
	}

	DEFINE(mallocfreetest, LOCATION_THREADS)
     DEFINE(copytest, LOCATION_THREADS)
     DEFINE(filltest, LOCATION_THREADS)
	DEFINE(cudaextracttest, LOCATION_THREADS)
	DEFINE(cudainserttest, LOCATION_THREADS)
	DEFINE(cudablockmovetest, LOCATION_THREADS)
	DEFINE(overlaptest, LOCATION_THREADS)
	DEFINE(sourceindextest,LOCATION_THREADS)
	DEFINE(sizetest,LOCATION_THREADS)
};
template <Region M>
void LocationTest<M>::sizetest() {
     size_t size = policy.free_memory();
     EXPECT_GT(size,0);

     size_t maximum = policy.max_memory();
     EXPECT_GT(maximum,0);
	
	int gpus = policy.number_of_gpus();
     EXPECT_GT(gpus,0);
};

template <Region M>
void LocationTest<M>::mallocfreetest() {
     void* p = NULL;
     p = policy.New(10);
     policy.Delete(p);
     EXPECT_TRUE(p);
};

template <Region M>
void LocationTest<M>::copytest() {
     int a = 1;
     int* a_ptr = &a;
     int b = 0;
     int* b_ptr = &b;

     location<Region::host>::MemCopy(a_ptr, b_ptr, sizeof(int));
     EXPECT_EQ(1, b);
};

template <>
void LocationTest<Region::device>::copytest() {

     size_t size = sizeof(int);
     int a = 1;
     int* a_d = static_cast<int*>(policy.New(size));
     ASSERT_TRUE(a_d);

     int b = 0;
     int* b_d = static_cast<int*>(policy.New(size));
     ASSERT_TRUE(b_d);

     hipMemcpy(a_d, &a, size, hipMemcpyHostToDevice);
     hipMemcpy(b_d, &b, size, hipMemcpyHostToDevice);

     location<Region::device>::MemCopy(a_d, b_d, 1);
     hipMemcpy(&b, b_d, size, hipMemcpyDeviceToHost);
     EXPECT_EQ(1, b);
};

template <Region M>
void LocationTest<M>::filltest() {
	int locallength=10;
	int size = locallength*sizeof(int);
	int value=2;
     int* a_d = static_cast<int*>(policy.New(size) );
     int* a_h = static_cast<int*>(std::malloc(size) );

	policy.fill_in(a_d, locallength,value);
	policy.MemCopy(a_d,a_h,size);
	for(int i=0; i<locallength; i++){
		EXPECT_EQ(a_h[i],value);
	}
	policy.Delete(a_d);
	std::free(a_h);

}

template <Region M>
void LocationTest<M>::overlaptest() {
	int offset=2;
	int locallength=length-offset;
	int localsize=locallength*sizeof(test_type);
		
	policy.MemCopy(	d_ptr+offset,	d_ptr,	locallength);
	hipMemcpy(		h_ptr,		d_ptr,	localsize,hipMemcpyDeviceToHost); 
	for(int i=0; i<locallength;i++){
		EXPECT_EQ(h_ptr[i],i+offset);
	}
	
	policy.MemCopy(	d_ptr,	d_ptr+offset,	locallength);
	hipMemcpy(		h_ptr,		d_ptr,	localsize,hipMemcpyDeviceToHost); 
	for(int i=offset; i<locallength;i++){
		EXPECT_EQ(h_ptr[i],i);
	}
}

template <>
void LocationTest<Region::host>::overlaptest() {
	int offset=2;
	int locallength=length-offset;
	int localsize=locallength*sizeof(test_type);
	
	policy.MemCopy(h_ptr+offset,h_ptr,localsize);

	for(int i=0; i<locallength;i++){
		EXPECT_EQ(h_ptr[i],i+offset);
	}
}

template <Region M>
void LocationTest<M>::cudaextracttest() {
	pointer tmp;
	int block=4;
	int offset=2;
	int tmp_size=size;
	hipMalloc((void**)&tmp,tmp_size); 
	cuda_overlapextract<pointer>
		<<<1,32>>>(	d_ptr,
						tmp,
						block,
						offset,
						length);
	hipMemcpy(h_ptr, tmp, size, hipMemcpyDeviceToHost);	
	
	int results[6]={4,5,8,9,12,13};
	for(int i=0; i<6;i++){
		EXPECT_EQ(results[i],h_ptr[i]);
	}
	hipFree(tmp);
}

template <Region M>
void LocationTest<M>::cudainserttest() {
	pointer tmp;
	int block=4;
	int offset=2;
	const int tmp_size=6;
	hipMalloc((void**)&tmp,tmp_size*sizeof(int)); 
	gpuErrorCheck( hipGetLastError()); 

	int tmp_h[tmp_size]={4,5,8,9,12,13};
	pointer tmp_h_ptr=tmp_h;  
	hipMemcpy(	tmp, 
				tmp_h_ptr, 
				sizeof(int)*tmp_size, 
				hipMemcpyDefault);
	gpuErrorCheck( hipGetLastError()); 

	cuda_overlapinsert<pointer>
		<<<1,32>>>(		d_ptr,
						tmp,
						block,
						offset,
						length);
	gpuErrorCheck( hipGetLastError()); 
	hipMemcpy(	h_ptr, 
				d_ptr, 
				sizeof(int)*tmp_size, 
				hipMemcpyDeviceToHost);	

	gpuErrorCheck( hipGetLastError()); 
//	int results[6]={4,5,8,9,12,13};
//	int indexes[6]={2,3,6,7,10,11};
//	for(int i=0; i<6;i++){
//		EXPECT_EQ(
//				results[i],
//				h_ptr[ indexes[i]]
//			);
//	}
//	^^^test no writen right,	
	gpuErrorCheck( hipGetLastError()); 	
	hipFree(tmp); 
	gpuErrorCheck( hipGetLastError()); 
}

template <Region M>
void LocationTest<M>::cudablockmovetest() {
	pointer tmp;
	int block=4;
	int offset=2;
	int tmp_size=size;
	hipMalloc((void**)&tmp,tmp_size); 
	hipMemcpy(tmp, d_ptr, tmp_size, hipMemcpyHostToDevice);
	cuda_blockmove<pointer,test_type>
		<<<1,32,block>>>(		d_ptr,
							tmp,
							block,
							offset,
							length);

	hipMemcpy(h_ptr, d_ptr, tmp_size, hipMemcpyDeviceToHost);	
	int results[16]={2,3,2,3,6,7,6,7,10,11,10,11,14,15,14,15};
	for(int i=0; i<length;i++){
		EXPECT_EQ(h_ptr[i],results[i]); 
	}
	hipFree(tmp); 
}

template <Region M>
void LocationTest<M>::sourceindextest() {
	#define NUMOFTEST_SIDT 12
	int param[NUMOFTEST_SIDT][3];
	for(int i=0; i<NUMOFTEST_SIDT; i++){
		param[i][0]=i; 
		param[i][1]=4;
		param[i][2]=2;
	};
	int result;
	int anwsers[NUMOFTEST_SIDT]={4,5,8,9,12,13,16,17,20,21,24,25};
	for(int i=0; i<NUMOFTEST_SIDT; i++){
		result=getSourceIndex<OVERLAP>(param[i][0],param[i][1],param[i][2]);
		EXPECT_EQ(result,anwsers[i]);
	}

	int block=5;
	int off=2;
	int anwsers2[NUMOFTEST_SIDT]={2,3,4,7,8,9,12,13,14,17,18,19};
	for(int i=0; i<NUMOFTEST_SIDT; i++){
		EXPECT_EQ(anwsers2[i],getSourceIndex<BLOCK>(i,block,off) );
	}
}
const Region host=Region::host; 
const Region unified=Region::unified; 
const Region pinned=Region::pinned; 
const Region device=Region::device; 
// python:key:policy=host unified device pinned
// python:key:tests=sizetest sourceindextest cudaextracttest cudainserttest cudablockmovetest overlaptest copytest mallocfreetest filltest
// python:key:concurrency=Single
// python:template=TEST_F($LocationTest<|policy|>$,|tests||concurrency|){this->|tests||concurrency|();}
// python:start
// python:include=location.test
#include "location.test"
// python:end

#undef LOCATION_THREADS
