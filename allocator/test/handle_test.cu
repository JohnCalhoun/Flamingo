#include "hip/hip_runtime.h"
// free_list_test.cu
#include <Handle.cpp>
#include <location.cu>
#include <gtest/gtest.h>

#include "MacroUtilities.cpp"

#include <cstddef>
#include <type_traits>
#define HANDLE_THREADS 8

using namespace Flamingo::Memory;

class HandleTest : public ::testing::Test {

	protected:
	virtual void SetUp() {
          int offset = 0;
          std::size_t size = 8;
          Handle<int> handle_int(offset, size, base_ptr);
          handle = handle_int;
     }
     int base[2] = {0, 1};
     int* base_ptr = base;
     Handle<int> handle;

     DEFINE(BuddyOffSetTest, HANDLE_THREADS)
     DEFINE(CopyConstructorTest, HANDLE_THREADS)
     DEFINE(JoinOperatorTest, HANDLE_THREADS)
     DEFINE(DereferenceOperatorTest, HANDLE_THREADS)
     DEFINE(IndirectionOperatorTest, HANDLE_THREADS)
     DEFINE(EqualityComparableTest, HANDLE_THREADS)
     DEFINE(DefaultConstructionTest, HANDLE_THREADS)
     DEFINE(CopyAssignableTest, HANDLE_THREADS)
     DEFINE(NullablePointerTest, HANDLE_THREADS)
     DEFINE(BiderectionalTest, HANDLE_THREADS)
     DEFINE(RandomAccessTest, HANDLE_THREADS)
     DEFINE(VoidTest, HANDLE_THREADS)
     DEFINE(BoolConvertTest, HANDLE_THREADS)
     DEFINE(IfStatementTest, HANDLE_THREADS)
     DEFINE(ConvertToConstTest, HANDLE_THREADS)
     DEFINE(ConstDereferenceTest, HANDLE_THREADS)
     DEFINE(DeviceSingleTest, HANDLE_THREADS)
     DEFINE(DeviceMultipleTest, HANDLE_THREADS)
     DEFINE(MemCopyTest, HANDLE_THREADS)
	DEFINE(AssignementTest,HANDLE_THREADS)
	DEFINE(ConstTest,HANDLE_THREADS)
};
void HandleTest::ConstTest() {
	int A=10;
	const int* A_ptr=&A; 

     Handle<const int> h(A_ptr);
};
void HandleTest::ConstDereferenceTest() {
     const Handle<int> h(0, 0, base_ptr);
     *h;
};

void HandleTest::BuddyOffSetTest() {
     int buddy = handle.buddy_offset();

     EXPECT_EQ(8, buddy);
};

void HandleTest::CopyConstructorTest() {
     Handle<int> handle_new(handle);

	Handle<int> handle_null(NULL); 
	Handle<int> handle_null_copy(handle_null); 

	Handle<int>* handle_ptr=&handle_null; 
	Handle<int> handle_move(*handle_ptr); 
};
void HandleTest::JoinOperatorTest() {
     Handle<int> h_1(handle);
     Handle<int> h_2(handle);

     h_1._offset = 8;
     h_1.combine(h_2);
     EXPECT_EQ(h_1._offset, 0);
     EXPECT_EQ(h_1._size, 16);
};
void HandleTest::DereferenceOperatorTest() {
     *base = 0;
     EXPECT_EQ(*handle, 0);
     *base = 1;
     EXPECT_EQ(*handle, 1);
};
void HandleTest::IndirectionOperatorTest() {
     Handle<int>*  handle_ptr = &handle;
     int offset = handle_ptr->_offset;
     EXPECT_EQ(offset, 0);
};
void HandleTest::EqualityComparableTest() {
     Handle<int> handle_2(0, 2, base_ptr);
     bool test_1 = (handle_2 == handle);
     EXPECT_TRUE(test_1);

     Handle<int> handle_3(1, 2, base_ptr);
     bool test_2 = (handle_3 == handle);
     EXPECT_FALSE(test_2);

     double* base_ptr2 = new double;
     Handle<double> handle_4(0, 8, base_ptr2);
     bool test_3 = (handle_4 == handle);
     EXPECT_FALSE(test_3);
};

void HandleTest::AssignementTest(){
	Handle<int> A;
	Handle<int> B;

	A=B; 
}
void HandleTest::DefaultConstructionTest() {
     Handle<int> handle_2();
};
void HandleTest::CopyAssignableTest() {
     Handle<int> handle_2 = handle;
     bool test = (handle_2 == handle);
     EXPECT_TRUE(test);
};
void HandleTest::NullablePointerTest() {
     std::nullptr_t null;
     Handle<int> handle_2(null);
     bool test_1 = (handle_2 == null);
     EXPECT_TRUE(test_1);

     Handle<int> handle_3 = null;
     bool test_2 = (handle_3 == null);
     EXPECT_TRUE(test_2);

     EXPECT_FALSE(handle == null);
     EXPECT_FALSE(null == handle);
     EXPECT_TRUE(handle != null);
     EXPECT_TRUE(null != handle);
};
void HandleTest::BiderectionalTest() {
     Handle<int> handle_2 = handle;
     handle_2++;
     EXPECT_TRUE(handle_2 != handle);
     handle_2--;
     EXPECT_TRUE(handle_2 == handle);
     --handle_2;
     ++handle_2;
     *handle_2++;
     EXPECT_EQ(*handle_2, 1);
     *handle_2--;
     EXPECT_EQ(*handle_2, 0);
};
void HandleTest::RandomAccessTest() {
     Handle<int> handle_2 = handle;

     handle_2 += 1;
     EXPECT_EQ(*handle_2, 1);
     handle_2 -= 1;
     EXPECT_EQ(*handle_2, 0);

     Handle<int> handle_3 = handle + 1;
     EXPECT_EQ(*handle_3, 1);
     Handle<int> handle_4 = 1 + handle;
     EXPECT_EQ(*handle_4, 1);
     Handle<int> handle_5 = handle_4 - 1;
     EXPECT_EQ(*handle_5, 0);

     Handle<int> handle_6 = handle;
     handle_6 += 1;
     int dif = handle_6 - handle;
     EXPECT_EQ(dif, 1);
     EXPECT_TRUE(handle < handle_6);
     EXPECT_FALSE(handle > handle_6);
     EXPECT_TRUE(handle <= handle_6);
     EXPECT_FALSE(handle >= handle_6);

     EXPECT_EQ(handle[0], 0);
     EXPECT_EQ(handle[1], 1);
};

void HandleTest::VoidTest() { Handle_void handle_void; }

void HandleTest::BoolConvertTest() { 

     int base_l[2] = {0, 1};
     int* base_ptr_l = base_l;
     Handle<int> handle_1(base_ptr_l);

	if(handle_1){
		EXPECT_TRUE(true); 
	}else{
		EXPECT_TRUE(false); 
	}
	
	Handle<int> handle_2(NULL); 

	if(handle_2){
		EXPECT_TRUE(false); 
	}else{
		EXPECT_TRUE(true); 
	}
}

void HandleTest::IfStatementTest() {
     bool True = true;

     True ? handle : 0;
     !True ? handle : 0;
}

void HandleTest::ConvertToConstTest() {
     bool convertable = std::is_convertible<Handle<int>,  const Handle<int> >::value;
     EXPECT_TRUE(convertable);
     convertable = std::is_convertible<Handle<int>,  const Handle<int> >::value;
     EXPECT_TRUE(convertable);
};

__global__ void transferS(Handle<int> x, Handle<int> y) {
     *y = *x;
};
void HandleTest::DeviceSingleTest() {
     int x = 2;
     int y = 0;

     int* x_d;
     int* y_d;

     hipMalloc((void**)&x_d, sizeof(int));
     hipMalloc((void**)&y_d, sizeof(int));

     hipMemcpy(x_d, &x, sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(y_d, &y, sizeof(int), hipMemcpyHostToDevice);

     Handle<int> x_h(0, 1, x_d);
     Handle<int> y_h(0, 1, y_d);
     transferS << <1, 1>>> (x_h, y_h);
     hipDeviceSynchronize();
     hipMemcpy(&y, y_d, sizeof(int), hipMemcpyDeviceToHost);
     EXPECT_EQ(x, y);

	Handle<int> copy(x_h); 
     hipFree(x_d);
     hipFree(y_d);
}

__global__ void transferM(Handle<int> x, Handle<int> y) {
     y[1] = x[1];
};

void HandleTest::DeviceMultipleTest() {
     int x[2] = {2, 3};
     int y[2] = {0, 1};

     int* x_d;
     int* y_d;
     int size = 2 * sizeof(int);

     hipMalloc((void**)&x_d, size);
     hipMalloc((void**)&y_d, size);
     hipMemcpy(x_d, &x, size, hipMemcpyHostToDevice);
     hipMemcpy(y_d, &y, size, hipMemcpyHostToDevice);

     Handle<int> x_h(0, 2, x_d);
     Handle<int> y_h(0, 2, y_d);
     transferM << <1, 1>>> (x_h, y_h);
     hipDeviceSynchronize();
     hipMemcpy(&y, y_d, size, hipMemcpyDeviceToHost);
     EXPECT_EQ(x[1], y[1]);
     EXPECT_FALSE(x[0] == y[0]);
     hipFree(x_d);
     hipFree(y_d);
}

template<Region SRC,Region DST>
void copyfunction(){
     typedef location<SRC>	Src_Location;
	typedef location<DST>	Dst_Location;

	Src_Location	src_location;
	Dst_Location	dst_location;

     int size = 3 * sizeof(int);

     int* x_d = static_cast<int*>(src_location.New(size));
     int* y_d = static_cast<int*>(dst_location.New(size));

     Handle<int> x_h(x_d);
     Handle<int> y_h(y_d);

     Src_Location::MemCopy(x_h, y_h, 3*sizeof(int) );

     src_location.Delete(x_d);
     dst_location.Delete(y_d);
}
void HandleTest::MemCopyTest() {
	copyfunction<Region::host,Region::host>();
	copyfunction<Region::host,Region::device>(); 
	copyfunction<Region::host,Region::pinned>();
	copyfunction<Region::host,Region::unified>();

	copyfunction<Region::device,Region::host>();
	copyfunction<Region::device,Region::device>();
	copyfunction<Region::device,Region::pinned>();
	copyfunction<Region::device,Region::unified>();

	copyfunction<Region::pinned,Region::host>();
	copyfunction<Region::pinned,Region::device>();
	copyfunction<Region::pinned,Region::pinned>();
	copyfunction<Region::pinned,Region::unified>();

	copyfunction<Region::unified,Region::host>();
	copyfunction<Region::unified,Region::device>();
	copyfunction<Region::unified,Region::pinned>();
	copyfunction<Region::unified,Region::unified>();
}

// python:key:function=ConstTest AssignementTest MemCopyTest DeviceSingleTest DeviceMultipleTest ConstDereferenceTest ConvertToConstTest IfStatementTest BoolConvertTest VoidTest BuddyOffSetTest CopyConstructorTest JoinOperatorTest DereferenceOperatorTest IndirectionOperatorTest EqualityComparableTest DefaultConstructionTest CopyAssignableTest NullablePointerTest BiderectionalTest RandomAccessTest
// python:key:concurency=Single Threaded
// python:template=TEST_F(HandleTest,|function||concurency|){this->|function||concurency|();};
// python:start
// python:include=handle.test
#include "handle.test"
// python:end
#undef HANDLE_THREADS
